#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernel_file.hpp"

__global__ void kernel_function() {
  printf("sizeof size_t from gpu : %ld\n", sizeof(size_t));
  printf("sizeof int from gpu : %ld\n", sizeof(int));
}

void run_kernel_function() {
  kernel_function<<<1, 1>>>();
  CUDA_ERROR;
  hipDeviceSynchronize();
  CUDA_ERROR;
}
